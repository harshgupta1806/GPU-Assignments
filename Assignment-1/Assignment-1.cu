#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;



// Task 1
__global__
void CalculateHadamardProduct(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    long long unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N*N)
    {
      unsigned int col = id% N;  
      unsigned int row = id / N;
      unsigned int IDA = N * row + col;
      unsigned int IDB = N * col + row;
      A[IDA] = A[IDA] * B[IDB];
    }

}

// Task 2
__global__
void FindWeightMatrix(long int* A, long int* B, int N) {
    long long unsigned id = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
     if(id < N*N)
    {
      if(A[id] < B[id])
        A[id] = B[id];
    }

}

//Task 3
__global__
void CalculateFinalMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
      unsigned int blockNo = gridDim.x * blockIdx.x + blockIdx.y;
      unsigned int noOfThreadsCrossed = blockNo * (blockDim.x * blockDim.y);
      unsigned int currThreadInBlock = noOfThreadsCrossed +  (blockDim.x * threadIdx.x + threadIdx.y);
      unsigned int id = currThreadInBlock; // Unique ID

    if(id < 4*N*N)
    {
      unsigned int col = id% (2*N);
      unsigned int row = id / (2*N);
      // printf("%d %d %d\t\t",id, row, col);
      unsigned int row_ = row%N;
      unsigned int col_ = col%N;
      unsigned int id1 = row_ * N + col_;
      B[id] = B[id] * A[id1];
    }
}


int main(int argc, char** argv) {

    // cout << "Enter N :: ";
    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }



    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);

    hipMalloc(&d_A, sizeof(long int) * N*N); // Allocating memory on GPU for Array A
    hipMalloc(&d_B, sizeof(long int) * N*N); // Allocating memory on GPU for Array B
 
    hipMemcpy(d_A, A, sizeof(long int) * N*N, hipMemcpyHostToDevice); // Copying array A on GPU
    hipMemcpy(d_B, B, sizeof(long int) * N*N, hipMemcpyHostToDevice); // Copying array B on GPU

    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;

    hipFree(d_B); // Memory Free from GPU for Array B




    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);


    hipMalloc(&d_C, sizeof(long int) * N*N); // Allocating memory on GPU for Array C
    hipMemcpy(d_C, C, sizeof(long int) * N*N, hipMemcpyHostToDevice); // Copying Array C on GPU

    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N); 
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;

    hipFree(d_C); // Memory Free for Array C on GPU
// Task 3
    hipMalloc(&d_D, sizeof(long int) *  4*N*N); // Allocating memory on GPU for Array C
    hipMemcpy(d_D, D, sizeof(long int) * 4*N*N, hipMemcpyHostToDevice); // Copying Array C on GPU

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);

    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;
    hipMemcpy(A, d_A, sizeof(long int) * N*N, hipMemcpyDeviceToHost);

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    hipFree(d_A); // Memory Free
    hipFree(d_D); // Memory Free 

    std::ofstream file("cuda.out");
     if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

     std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}